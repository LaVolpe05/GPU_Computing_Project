#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hipsparse.h>
#include <math.h>


/*** A new type representing an Hybrid matrix***/
typedef struct{
    int cut_off;
    float * ell_data;
    int * ell_col;
    int ell_size;
    float * coo_data;
    int * coo_col;
    int * coo_row;
    int coo_size;
}Hyb;


/*** Input : Two matrices on Hybrid format 

    Output : Return 2 array ; one is the ELL values of the multiplication and the other one is the COO values of the multiplication 
***/


__global__ void HYB_multiplication(float * ell_data , int * ell_col_ids,int size_of_ell,int cut_off ,float * coo_data,int * col_ids,int * rows_ids, int size_of_coo , float *inELL, float * inCOO ,float * outEll , float * outCoo ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nbRow = size_of_ell/cut_off;
    //ELL multiplication
    if (idx < nbRow){
        int row = idx ;
        float dot = 0;
        for (int element = 0; element < cut_off ; element++){ //elements_in_rows
            int element_offset = row + element * nbRow;
            dot += ell_data[element_offset]* inELL[ell_col_ids[element_offset]];

        }
        atomicAdd(outEll + row, dot);

    }
    //COO multiplication
    for (int element = idx ; element < size_of_coo; element += blockDim.x * gridDim.x){
        float dot = coo_data[element] * inCOO[col_ids[element]];
        atomicAdd(outCoo+ rows_ids[element],dot);
    }
}
/*** Input : A matrix on Hybrid format composed of(Ell : values and siz ; COO :values and size ) plus a scalar we multiply it with .

    Output : Return a matrix in Hybrid format 

***/
__global__ void HYB_multiplication_scalar(float * ell_data ,int size_of_ell,float * coo_data, int size_of_coo , float scalar ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nbRow = size_of_ell/cut_off;
    //ELL multiplication with scalar
    if (idx < nbRow){
        for (int element = 0; element < cut_off ; element++){
            ell_data[element] = ell_data[element] * scalar;
        }

    }
    //COO multiplication with scalar
    for (int element = idx ; element < size_of_coo; element += blockDim.x * gridDim.x){
        coo_data[element]  = coo_data[element] * scalar;
    }
    
}


/*** Input : Two matrices on Hybrid format 

    Output : Return the first matrix with the addition performed with the second one 
    ***/
__global__ void HYB_addition(float * ell_data , int * ell_col_ids,int size_of_ell,int cut_off ,float * coo_data,int * col_ids, int size_of_coo , float *inELL, float * inCOO  ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nbRow = size_of_ell/cut_off;
    //ELL addition
    if (idx < nbRow){
        int row = idx ;
        for (int element = 0; element < cut_off ; element++){ //elements_in_rows
            int element_offset = row + element * nbRow;
            ell_data[element_offset]= ell_data[element_offset] + inELL[ell_col_ids[element_offset]];

        }
    }
    //COO addition
    for (int element = idx ; element < size_of_coo; element += blockDim.x * gridDim.x){
        coo_data[element] = coo_data[element] + inCOO[col_ids[element]];
    }
    
}t

/*** Input : 

    Output : A 2-Dimensional array , each rows is the list of index of the matrices to be multiply 
***/
int **getCombinaison(int start ,int end , int length , int dimension){
    int size = length-1 ;
    //int arr_size = pow(dimension,size);
    char arr[2][3] = {{11, 22, 33},{11, 22, 33}};
    int indices[size];
    int new_arr[dimension*dimension*size];
    for (int i = 0; i < dimension*dimension*size; i++){
        new_arr[i] = 0;
    }
    //printf("here");
    for (int i = 0; i < size; i++){
        indices[i] = 0;
    }
    //printf("here");
    while (1) {
    
        // print current combination
        for (int i = 0; i < size; i++){
            if(i != 0 && i != size-1){
                printf("%d",arr[i][indices[i]]);
            }else{
                if(i == 0){
                    printf("%d ",arr[i][indices[i]]+(start*100));
                }
                if(i == size-1){
                    printf("%d ",arr[i][indices[i]]*10+(end));
                }
            }
            
            //new_arr[i] = arr[i][indices[i]];
            //printf("%d ",new_arr[i]);
            
        }
        printf("\n");
 
        // find the rightmost array that has more
        // elements left after the current element
        // in that array
        int next = size - 1;
       // printf("%d , %d , %d \n", next,indices[next],dimension);
        while (next >= 0 && (indices[next] + 1 >= dimension)){
            next--;
        }
        // no such array is found so no more
        // combinations left
        if (next < 0){
            return;
        }
        // if found move to next element in that
        // array
        indices[next]++;
 
        // for all arrays to the right of this
        // array current index again points to
        // first element
        for (int i = next + 1; i < size; i++){
            indices[i] = 0;
        }
    }
     /*for (int i = 0; i < dimension*dimension; i++){
        for (int j = 0; j < size; j++){
        printf("%d ",new_arr[i][indices[i]]);
    }
    printf("\n");
    }*/
        
    
}
    

    
    return arr;
}
/*** Input : The multiplex starting and ending , the length of the path , the SupraMatrix dimension (a squared matrix ) ,
 the combinnaison which conduct the matrix mutlpilication 


    Output : Return a matrix on Hybrid format 
***/
__global__ void getMatrixFromCombinaison(int start ,int end , int length , int dimension, HYB * SupraMatrix, int * combinaison  ){

    


}

/*** Intput : cut_off value , the COO structure size , the rows array of the COO structure

    Output : Return the size of the new COO structure for the COOToHybrid transformation
***/

int size_of_new_coo(int * rows, int COO_size , int cut_off){
    int size = 0;
    int count = 0;
    for (int i = 0 ; i < COO_size; i++){
        if (i == 0){
            count++;
        }else{
            if( rows[i-1] == rows[i]){
                count++;
            }else{
                count = 1;
            }
        }
        if (count > cut_off){
            size++;
        }
    }
    return size;
}

/*** Input : A COO structure composed of (values , columns , rows and the size of the arrays), the dimension of the original matrix , the value of the cut_off,
    a pre-composed hybrid structure composed of (Ell : values and columns ; COO :values , columns , rows)

    Output : An Hybrid structure composed of (Ell : values and columns ; COO :values , columns , rows)
***/

int cooToHyb(int* values,int * columns, int * rows ,int COO_size , int nbCol , int nbRow,int cut_off,int ** ELL_Values, int ** ELL_Indexes,int ** COO_Values, int ** COO_Col , int ** COO_Row){
  int size_of_coo = size_of_new_coo(rows,COO_size,cut_off);
  int size_of_ell = nbRow*cut_off;
  int temp_COO_values[size_of_coo];
  int temp_COO_row[size_of_coo];
  int temp_COO_col[size_of_coo];
  int temp_ELL_values[size_of_ell];
  int temp_ELL_indexes[size_of_ell];
  int elements_in_the_row = 0 ; 
  int current_row = 0;
  int index_ELL = 0;
  int index_COO = 0;
  for (int i = 0 ; i < COO_size ; i++){
      if(rows[i] == current_row ){
          //ELL format if we are below the cut-off 
          if(elements_in_the_row < cut_off){
              temp_ELL_values[index_ELL] = values[i];
              temp_ELL_indexes[index_ELL] = columns[i];
              //printf("ELL value : %d and columns %d \n",temp_ELL_values[index_ELL],temp_ELL_indexes[index_ELL]);
              index_ELL++;
          }else{// COO format if we are above the cut-off
                temp_COO_values[index_COO] = values[i];
                temp_COO_col[index_COO] = columns[i];
                temp_COO_row[index_COO] = rows[i];
                //printf("COO value : %d and columns %d and row %d  \n",temp_COO_values[index_COO],temp_COO_col[index_COO],temp_COO_row[index_COO]);
                index_COO++;
          }
          elements_in_the_row++;
          
      }else{
          //Handle with rows fill with zero
          if((rows[i]-current_row) != 1){
            for(int j = 0 ; j < (cut_off*(rows[i]-current_row-1));j++){
                temp_ELL_values[index_ELL] = -1 ;
                temp_ELL_indexes[index_ELL] = -1;
                //printf("ELL(fill_line) value : %d and columns %d \n",temp_ELL_values[index_ELL],temp_ELL_indexes[index_ELL]);
                index_ELL++;
            }
            

          }
          //Complete line of ELL format
          if (elements_in_the_row < cut_off){
                for(int j = 0 ; j < (cut_off-elements_in_the_row);j++){
                    temp_ELL_values[index_ELL] = -1 ;
                    temp_ELL_indexes[index_ELL] = -1;
                    //printf("ELL(complete) value : %d and columns %d\n",temp_ELL_values[index_ELL],temp_ELL_indexes[index_ELL]);
                    index_ELL++;
                }

          }
          current_row = rows[i];
          temp_ELL_values[index_ELL] = values[i];
          temp_ELL_indexes[index_ELL] = columns[i];
          //printf("ELL(new line) value : %d and columns %d   \n",temp_ELL_values[index_ELL],temp_ELL_indexes[index_ELL]);
          index_ELL++;
          elements_in_the_row = 1;


      }
    
  }
  printf("Actualise ! \n");
  *ELL_Values = temp_ELL_values;
  *ELL_Indexes = temp_ELL_indexes;
  *COO_Values = temp_COO_values;
  *COO_Col = temp_COO_col;
  *COO_Row = temp_COO_row;
  return index_ELL;
}

void Katz_Similarity(int theta ,int pathLength,int start,int end){
    int * temp_COO_values;
    int * temp_COO_row;
    int * temp_COO_col;
    int * temp_ELL_values;
    int * temp_ELL_indexes;
    dim3 dimBlock( RowA/cut_off, RowA/cut_off);
	dim3 dimGrid( 1, 1 );
    
    for(int k = 0 ; k < pathLength ; k++){
        HYB_addition<<<dimGrid,dimBlock>>>(temp_COO_values,temp_COO_row,temp_COO_col,temp_ELL_values,temp_ELL_indexes,(matrixFactorMultiplication(pow(theta,k),getMatrixFromCombinaison(start,end,k,dimension,HYB supraAdjacency,combination))));
    }
Impl



}










int main()
{



   /* int COO_size_MatriceA = 10;
    int Row_size_MatriceA = 5;
    int Col_size_MatriceA = 5;
    int cut_off_MatriceA = 2;

    int valueMatA[COO_size_MatriceA]=  {1,5,8,2,3,8,1,4,5,6};
    int colMatA[COO_size_MatriceA] = {0,0,1,2,0,0,1,0,3,4};
    int rowMatA[COO_size_MatriceA] = {0,1,1,1,2,3,3,4,4,4};


    int * ELL_Values_MatriceA;
    int * ELL_Indexes_MatriceA;
    int * COO_Values_MatriceA;
    int * COO_Col_MatriceA;
    int * COO_Row_MatriceA;

    
    
    
    int size_of_coo_MatriceA = size_of_new_coo(rowMatA,COO_size_MatriceA,cut_off_MatriceA);
    int size_ELL_MatriceA = cooToHyb(valueMatA,colMatA,rowMatA,COO_size_MatriceA,Col_size_MatriceA,Row_size_MatriceA,cut_off_MatriceA,&ELL_Values_MatriceA,&ELL_Indexes_MatriceA,&COO_Values_MatriceA,&COO_Col_MatriceA,&COO_Row_MatriceA);

    /*** Printing result of COO to HYB format ***//*
   printf("The ELL part of the sparse format is :\n");
    printf("|| Values | Indexes ||\n");
    for (int i = 0 ; i < size_ELL_MatriceA;i++){
        printf("|| %d | %d ||\n",*(ELL_Values_MatriceA+i),*(ELL_Indexes_MatriceA+i));
    }
    printf("The COO part of the sparse format is :\n");
    printf("|| Values | Columns | Rows ||\n");
    for (int i = 0 ; i < size_of_coo_MatriceA;i++){
        printf("|| %d | %d | %d ||\n",*(COO_Values_MatriceA+i),*(COO_Col_MatriceA+i),*(COO_Row_MatriceA+i));
    }*/
    //int size = 840058;
    //float *mypointer =(float*) 2914545049664;
    //printf("%f",*(mypointer));
    /*for (int i = 0 ; i < size ; i++){
        printf("%f ",*(mypointer + (i*sizeof(float))));
    }*/





    /*** TEST Hybrid Multiplication Kernel***/
    int cut_off = 2;
    /*****************************************************/
    /*int MatriceA[16] = {5,6,7,8,
                        1,2,3,4,
                        9,10,11,12,
                        0,0,1,0};
     int ColA = 4;*/int RowA = 4 ;
    int ELL_ValuesA[cut_off*RowA] ={ 5,6,
                                     1,2,
                                     9,10,
                                     1,-1};
    int ELL_Col_idsA[cut_off*RowA] ={ 0,1,
                                      0,1,
                                      0,1,
                                      2,-1};
    int Coo_ValuesA[6] = {7,8,3,4,11,12};
    int Coo_ColA[6] = {2,3,2,3,2,3};
    int Coo_RowA[6] = {0,0,1,1,2,2};


    int ell_sizeA = cut_off*RowA*sizeof(int);
    int *cuda_Ell_Val_A;
    int *cuda_Ell_Col_A;
    hipMalloc( (void**)&cuda_Ell_Val_A, ell_sizeA );
    hipMalloc( (void**)&cuda_Ell_Col_A, ell_sizeA );
    hipMemcpy( cuda_Ell_Val_A, &ELL_ValuesA, ell_sizeA, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Ell_Col_A, &ELL_Col_idsA, ell_sizeA, hipMemcpyHostToDevice ); 

    int coo_sizeA = 6*sizeof(int);
    int *cuda_Coo_Val_A;
    int *cuda_Coo_Col_A;
    int *cuda_Coo_Row_A;
    hipMalloc( (void**)&cuda_Coo_Val_A, coo_sizeA );
    hipMalloc( (void**)&cuda_Coo_Col_A, coo_sizeA );
    hipMalloc( (void**)&cuda_Coo_Row_A, coo_sizeA );
    hipMemcpy( cuda_Coo_Val_A, &Coo_ValuesA, coo_sizeA, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Coo_Col_A, &Coo_ColA, coo_sizeA, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Coo_Row_A, &Coo_RowA, coo_sizeA, hipMemcpyHostToDevice ); 

    /*****************************************************/
    /*int MatriceB[16] = {5,0,0,8,
                        2,4,5,0,
                        6,0,0,0,
                        7,8,0,0};
    int ColB = 4; */int RowB = 4 ;
    int ELL_ValuesB[cut_off*RowB] ={ 5,8,
                                    2,4,
                                    6,-1,
                                    7,8};
    int ELL_Col_idsB[cut_off*RowB] ={ 0,3,
                                      0,1,
                                      0,-1,
                                      0,1};
    int Coo_ValuesB[1] = {5};
    int Coo_ColB[1] = {2};
    int Coo_RowB[1] = {1};


    int ell_sizeB = cut_off*RowB*sizeof(int);
    int *cuda_Ell_Val_B;
    int *cuda_Ell_Col_B;
    hipMalloc( (void**)&cuda_Ell_Val_B, ell_sizeB );
    hipMalloc( (void**)&cuda_Ell_Col_B, ell_sizeB );
    hipMemcpy( cuda_Ell_Val_B, &ELL_ValuesB, ell_sizeB, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Ell_Col_B, &ELL_Col_idsB, ell_sizeB, hipMemcpyHostToDevice );
     

    int coo_sizeB = sizeof(int);
    int *cuda_Coo_Val_B;
    int *cuda_Coo_Col_B;
    int *cuda_Coo_Row_B;
    hipMalloc( (void**)&cuda_Coo_Val_B, coo_sizeA );
    hipMalloc( (void**)&cuda_Coo_Col_B, coo_sizeA );
    hipMalloc( (void**)&cuda_Coo_Row_B, coo_sizeA );
    hipMemcpy( cuda_Coo_Val_B, &Coo_ValuesB, coo_sizeB, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Coo_Col_B, &Coo_ColB, coo_sizeB, hipMemcpyHostToDevice ); 
    hipMemcpy( cuda_Coo_Row_B, &Coo_RowB, coo_sizeB, hipMemcpyHostToDevice ); 
    /*****************************************************/
    int size_output = 16;
    int * cudaoutELL;
    int * cudaoutCOO ;
    int outELL[size_output];
    int outCOO[size_output];
    

    hipMalloc( (void**)&cudaoutELL, size_output*sizeof(int) );
    hipMalloc( (void**)&cudaoutCOO, size_output*sizeof(int) );
    
    
    
    /*****************************************************/

    dim3 dimBlock( RowA/cut_off, RowA/cut_off);
	dim3 dimGrid( 1, 1 );
    
    HYB_multiplication<<<dimGrid, dimBlock>>>( cuda_Ell_Val_A , cuda_Ell_Col_A,cut_off*RowA,cut_off ,cuda_Coo_Val_A,cuda_Coo_Col_A,cuda_Coo_Row_A, 6 ,  cuda_Ell_Val_B ,  cuda_Coo_Val_B ,cudaoutELL , cudaoutCOO);
    hipDeviceSynchronize();
    hipMemcpy( outELL, &cudaoutELL, coo_sizeB, hipMemcpyDeviceToHost ); 
    hipMemcpy( outCOO, &cudaoutCOO, coo_sizeB, hipMemcpyDeviceToHost ); 
    printf("ELL");
    for (int i = 0 ; i < size_output ; i++){
        printf("%d ",outELL[i]);
        
    }
    printf("\nCOO");
    for (int i = 0 ; i < size_output ; i++){
        printf("%d ",outCOO[i] );
    }
    

}
